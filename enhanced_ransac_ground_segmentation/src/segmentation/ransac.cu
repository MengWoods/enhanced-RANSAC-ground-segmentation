#include "hip/hip_runtime.h"
/*
 * Enhanced RANSAC Ground Segmentation
 * Copyright (c) 2025 Menghao Woods
 *
 * Licensed under the MIT License. See LICENSE file in the project root for details.
 */
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <pcl/io/pcd_io.h>
#include <pcl/point_types.h>

#include <cmath>

#define THREADS_PER_BLOCK 256

using Point = pcl::PointXYZI;
using PointCloud = pcl::PointCloud<pcl::PointXYZI>;
using PointCloudPtr = pcl::PointCloud<pcl::PointXYZI>::Ptr;

/**
 * @brief Kernel to compute RANSAC plane fitting.
 */
__global__ void ransacKernel(const Point *points, int num_points, int max_iterations,
                             float distance_threshold, float *best_plane)
{
    __shared__ float best_model[4];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Random seed
    hiprandState state;
    hiprand_init(clock64(), tid, 0, &state);

    float best_inlier_count = 0;

    for (int i = 0; i < max_iterations; ++i)
    {
        // Randomly select 3 points
        int idx1 = hiprand(&state) % num_points;
        int idx2 = hiprand(&state) % num_points;
        int idx3 = hiprand(&state) % num_points;

        // Avoid duplicate points
        if (idx1 == idx2 || idx2 == idx3 || idx1 == idx3)
            continue;

        Point p1 = points[idx1];
        Point p2 = points[idx2];
        Point p3 = points[idx3];

        // Compute plane normal
        float ux = p2.x - p1.x, uy = p2.y - p1.y, uz = p2.z - p1.z;
        float vx = p3.x - p1.x, vy = p3.y - p1.y, vz = p3.z - p1.z;

        float a = uy * vz - uz * vy;
        float b = uz * vx - ux * vz;
        float c = ux * vy - uy * vx;
        float d = -(a * p1.x + b * p1.y + c * p1.z);

        // Normalize
        float norm = sqrt(a * a + b * b + c * c);
        a /= norm;
        b /= norm;
        c /= norm;
        d /= norm;

        // Count inliers
        int inlier_count = 0;
        for (int j = 0; j < num_points; ++j)
        {
            float dist = fabs(a * points[j].x + b * points[j].y + c * points[j].z + d);
            if (dist < distance_threshold)
                inlier_count++;
        }

        // Update best model
        if (inlier_count > best_inlier_count)
        {
            best_inlier_count = inlier_count;
            best_model[0] = a;
            best_model[1] = b;
            best_model[2] = c;
            best_model[3] = d;
        }
    }

    // Save best model
    if (tid == 0)
    {
        for (int i = 0; i < 4; ++i)
            best_plane[i] = best_model[i];
    }
}

/**
 * @brief Wrapper function to launch CUDA kernel.
 */
void ransacFitPlaneGPU(const Point *d_points, int num_points, int max_iterations,
                       float distance_threshold, float *d_plane_coeffs)
{
    int num_blocks = (num_points + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    ransacKernel<<<num_blocks, THREADS_PER_BLOCK>>>(d_points, num_points, max_iterations, distance_threshold, d_plane_coeffs);

    hipDeviceSynchronize();
}
